#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#define SIZE 1024 * 1024

__global__ void vectorAdditionKernel(double* A, double* B, double* C, int arraySize) {
    // Get thread ID.
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if thread is within array bounds.
    if (threadID < arraySize) {
        // Add a and b.
        C[threadID] = A[threadID] + B[threadID];
    }
}

/**
 * Wrapper function for the CUDA kernel function.
 * @param A Array A.
 * @param B Array B.
 * @param C Sum of array elements A and B directly across.
 * @param arraySize Size of arrays A, B, and C.
 */
void kernel(double* A, double* B, double* C, int arraySize) {

    // Initialize device pointers.
    double* d_A, * d_B, * d_C;

    // Allocate device memory.
    hipMalloc((void**)&d_A, arraySize * sizeof(double));
    hipMalloc((void**)&d_B, arraySize * sizeof(double));
    hipMalloc((void**)&d_C, arraySize * sizeof(double));

    // Transfer arrays a and b to device.
    hipMemcpy(d_A, A, arraySize * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, arraySize * sizeof(double), hipMemcpyHostToDevice);

    // Calculate blocksize and gridsize.
    dim3 blockSize(512, 1, 1);
    dim3 gridSize(512 / arraySize + 1, 1);

    // Launch CUDA kernel.
    vectorAdditionKernel << <gridSize, blockSize >> > (d_A, d_B, d_C, arraySize);

    // Copy result array c back to host memory.
    hipMemcpy(C, d_C, arraySize * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main()
{
    double* a, * b, * c;
    a = (double*)malloc(SIZE * sizeof(double));
    b = (double*)malloc(SIZE * sizeof(double));
    c = (double*)malloc(SIZE * sizeof(double));
    for (int i = 0; i < SIZE; ++i)
    {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }
    //CALL CUDA to do the work
    kernel(a, b, c, SIZE);

    for (int i = 0; i < 10; ++i) {
        printf("c[%d]=%.0f\n", i, c[i]);
    }
    free(a);
    free(b);
    free(c);
    return 0;

}